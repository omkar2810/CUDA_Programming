#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"
#define TILE_WIDTH 16


__global__ void unrollAndMatrixMultiplyShared(const float *input_kernel, const float *input_images, float *output_images,
                                     int numARows, int numAColumns,
                                     int numBRows, int numBColumns,
                                     int numCRows, int numCColumns,
                                     int Batch, int K, int Channel, int Height, int Width) {
  #define in_4d(i3, i2, i1, i0) input_images[(i3) * (Channel * Height * Width) + (i2) * (Height * Width) + (i1) * (Width) + i0]

  __shared__ float subTileA[TILE_WIDTH][TILE_WIDTH];
  __shared__ float subTileB[TILE_WIDTH][TILE_WIDTH]; 

  int global_batch = blockIdx.z;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int width_tiles = ceil((1.0*numCColumns)/TILE_WIDTH);
  int row = blockIdx.y*blockDim.y + ty;
  int col = blockIdx.x*blockDim.x + tx; 
  float pval = 0.0f;
  for (int q = 0; q < (numAColumns-1)/TILE_WIDTH + 1; q++) 
  { 
    if ((row < numARows) && (q*TILE_WIDTH+tx < numAColumns))
      subTileA[ty][tx] = input_kernel[row*numAColumns + q*TILE_WIDTH+tx];
    else
      subTileA[ty][tx] = 0.0f;
    
    if ((col < numBColumns) && (q*TILE_WIDTH+ty < numBRows))
    {
      int h_unroll = (q*TILE_WIDTH+ty); // 6
      int w_unroll = col; // 1
      int channel = h_unroll/(K*K); // 1
      int Width_out = Width - K + 1;
      int h = w_unroll/Width_out; // 0
      int w = w_unroll%Width_out; // 1
      int p = (h_unroll - channel*K*K)/K; // (6-4)/2 = 1  
      int _q = (h_unroll - channel*K*K)%K; // (6-4)%2 = 0
      subTileB[ty][tx] = in_4d(global_batch, channel, h+p, w+_q);
    }
    else
      subTileB[ty][tx] = 0.0f;

    __syncthreads();
    for (int k = 0; k < TILE_WIDTH; k++)
      pval += subTileA[ty][k] * subTileB[k][tx];
    __syncthreads();

  }

  if ((row<numCRows)&&(col<numCColumns))
  {
    output_images[global_batch*(numCRows*numCColumns) + row*numCColumns+col] = pval;
  }

  #undef in_4d
}
	
__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // Allocate memory and copy over the relevant data structures to the GPU

    // We pass double pointers for you to initialize the relevant device pointers,
    //  which are passed to the other two functions.

    // Useful snippet for error checking
    // hipError_t error = hipGetLastError();
    // if(error != hipSuccess)
    // {
    //     std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
    //     exit(-1);
    // }
    int Height_out = Height - K + 1;
    int Width_out = Width - K + 1;
    int in_size = Batch*Channel*Height*Width;
    int out_size = Batch*Map_out*Height_out*Width_out;
    int mask_size = Map_out*Channel*K*K;

    hipMalloc((void **)device_input_ptr, in_size*sizeof(float));
    hipMalloc((void **)device_output_ptr, out_size*sizeof(float));
    hipMalloc((void **)device_mask_ptr, mask_size*sizeof(float));
    hipMemcpy(*device_input_ptr, host_input, in_size*sizeof(float), hipMemcpyHostToDevice);
    // hipMemcpy(*device_output_ptr, host_output, out_size*sizeof(float), hipMemcpyHostToDevice);    
    hipMemcpy(*device_mask_ptr, host_mask, mask_size*sizeof(float), hipMemcpyHostToDevice);    

}


__host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // Set the kernel dimensions and call the kernel
    int Height_out = Height - K + 1;
    int Width_out = Width - K + 1;
    int width_tiles = ceil((1.0*Width_out)/TILE_WIDTH);
    int height_tiles = ceil((1.0*Height_out)/TILE_WIDTH);

    int Width_unroll = Height_out*Width_out;
    int Height_unroll = K*K*Channel;
    int Height_kernel = Map_out;
    int Width_kernel = K*K*Channel;

    int Height_output = Height_kernel;
    int Width_output = Width_unroll;

    int unroll_size = Batch*Height_unroll*Width_unroll;

    dim3 gridDim(ceil(1.0*Width_output/TILE_WIDTH), ceil(1.0*Height_output/TILE_WIDTH), Batch);
    dim3 blockDim(TILE_WIDTH, TILE_WIDTH, 1);

    unrollAndMatrixMultiplyShared<<<gridDim, blockDim>>>(device_mask, device_input, device_output, Height_kernel, Width_kernel, Height_unroll, Width_unroll, Height_output, Width_output, Batch, K, Channel, Height, Width);

}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    int Height_out = Height - K + 1;
    int Width_out = Width - K + 1;
    int out_size = Batch*Map_out*Height_out*Width_out;
    // Copy the output back to host
    hipMemcpy(host_output, device_output, out_size*sizeof(float), hipMemcpyDeviceToHost);    

    // Free device memory
    hipFree(device_input);
    hipFree(device_output);
    hipFree(device_mask);

}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}