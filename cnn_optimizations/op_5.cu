#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"
#define TILE_WIDTH 16

__global__ void conv_forward_kernel(float * __restrict__ output, const float* __restrict__ input, const float * __restrict__ mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    /*
    Modify this function to implement the forward pass described in Chapter 16.
    We have added an additional dimension to the tensors to support an entire mini-batch
    The goal here is to be correct AND fast.

    Function paramter definitions:
    output - output
    input - input
    mask - convolution kernel
    Batch - batch_size (number of images in x)
    Map_out - number of output feature maps
    Channel - number of input feature maps
    Height - input height dimension
    Width - input width dimension
    K - kernel height and width (K x K)
    */

    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;
    int width_tiles = ceil((1.0*Width_out)/TILE_WIDTH);
    int m = blockIdx.x;
    int x_out = (blockIdx.y%width_tiles)*TILE_WIDTH + threadIdx.x;
    int y_out = (blockIdx.y/width_tiles)*TILE_WIDTH + threadIdx.y;
    int batch = blockIdx.z;
    if(x_out>=Width_out || y_out >= Height_out)
        return;   
    // We have some nice #defs for you below to simplify indexing. Feel free to use them, or create your own.
    // An example use of these macros:
    // float a = in_4d(0,0,0,0)
    // out_4d(0,0,0,0) = a

    #define out_4d(i3, i2, i1, i0) output[(i3) * (Map_out * Height_out * Width_out) + (i2) * (Height_out * Width_out) + (i1) * (Width_out) + i0]
    #define in_4d(i3, i2, i1, i0) input[(i3) * (Channel * Height * Width) + (i2) * (Height * Width) + (i1) * (Width) + i0]
    #define mask_4d(i3, i2, i1, i0) mask[(i3) * (Channel * K * K) + (i2) * (K * K) + (i1) * (K) + i0]

    // Insert your GPU convolution kernel code here
    float result = 0.0f;
    for(int c=0; c<Channel; c++)
    {
        for(int h=0; h<K; h++)
        {
            // for(int w=0; w<K; w+=2)
            // {
            //     result += in_4d(batch, c, y_out+h, x_out+w)*mask_4d(m, c, h, w);
            //     if(w+1 < K)
            //         result += in_4d(batch, c, y_out+h, x_out+w+1)*mask_4d(m, c, h, w+1);
            //     // result += in_4d(batch, c, y_out+h, x_out+w+2)*mask_4d(m, c, h, w+2);

            // }
            int w = 0;
            result += in_4d(batch, c, y_out+h, x_out+w)*mask_4d(m, c, h, w);
            result += in_4d(batch, c, y_out+h, x_out+w+1)*mask_4d(m, c, h, w+1);
            result += in_4d(batch, c, y_out+h, x_out+w+2)*mask_4d(m, c, h, w+2);
            result += in_4d(batch, c, y_out+h, x_out+w+3)*mask_4d(m, c, h, w+3);
            result += in_4d(batch, c, y_out+h, x_out+w+4)*mask_4d(m, c, h, w+4);
            result += in_4d(batch, c, y_out+h, x_out+w+5)*mask_4d(m, c, h, w+5);
            result += in_4d(batch, c, y_out+h, x_out+w+6)*mask_4d(m, c, h, w+6);
        }
    }
    out_4d(batch, m, y_out, x_out) = result;

    #undef out_4d
    #undef in_4d
    #undef mask_4d
}

	
__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // Allocate memory and copy over the relevant data structures to the GPU

    // We pass double pointers for you to initialize the relevant device pointers,
    //  which are passed to the other two functions.

    // Useful snippet for error checking
    // hipError_t error = hipGetLastError();
    // if(error != hipSuccess)
    // {
    //     std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
    //     exit(-1);
    // }
    int Height_out = Height - K + 1;
    int Width_out = Width - K + 1;
    int in_size = Batch*Channel*Height*Width;
    int out_size = Batch*Map_out*Height_out*Width_out;
    int mask_size = Map_out*Channel*K*K;
    hipMalloc((void **)device_input_ptr, in_size*sizeof(float));
    hipMalloc((void **)device_output_ptr, out_size*sizeof(float));
    hipMalloc((void **)device_mask_ptr, mask_size*sizeof(float));
    hipMemcpy(*device_input_ptr, host_input, in_size*sizeof(float), hipMemcpyHostToDevice);
    // hipMemcpy(*device_output_ptr, host_output, out_size*sizeof(float), hipMemcpyHostToDevice);    
    hipMemcpy(*device_mask_ptr, host_mask, mask_size*sizeof(float), hipMemcpyHostToDevice);    

}


__host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // Set the kernel dimensions and call the kernel
    int Height_out = Height - K + 1;
    int Width_out = Width - K + 1;
    int width_tiles = ceil((1.0*Width_out)/TILE_WIDTH);
    int height_tiles = ceil((1.0*Height_out)/TILE_WIDTH);
    int grid_z = Batch;
    int grid_y = width_tiles*height_tiles;
    int grid_x = Map_out;
    dim3 gridDim(grid_x, grid_y, grid_z);
    dim3 blockDim(TILE_WIDTH, TILE_WIDTH, 1);
    conv_forward_kernel<<<gridDim, blockDim>>>(device_output, device_input, device_mask, Batch, Map_out, Channel, Height, Width, K);

}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    int Height_out = Height - K + 1;
    int Width_out = Width - K + 1;
    int out_size = Batch*Map_out*Height_out*Width_out;
    // Copy the output back to host
    hipMemcpy(host_output, device_output, out_size*sizeof(float), hipMemcpyDeviceToHost);    

    // Free device memory
    hipFree(device_input);
    hipFree(device_output);
    hipFree(device_mask);

}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}