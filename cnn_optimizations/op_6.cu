#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"
#define TILE_WIDTH 16

__global__ void conv_forward_kernel(float *output, const float *input, const float *mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    /*
    Modify this function to implement the forward pass described in Chapter 16.
    We have added an additional dimension to the tensors to support an entire mini-batch
    The goal here is to be correct AND fast.

    Function paramter definitions:
    output - output
    input - input
    mask - convolution kernel
    Batch - batch_size (number of images in x)
    Map_out - number of output feature maps
    Channel - number of input feature maps
    Height - input height dimension
    Width - input width dimension
    K - kernel height and width (K x K)
    */

    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;
    int width_tiles = ceil((1.0*Width_out)/TILE_WIDTH);
    int m = blockIdx.x;
    int x_out = (blockIdx.y%width_tiles)*TILE_WIDTH + threadIdx.x;
    int y_out = (blockIdx.y/width_tiles)*TILE_WIDTH + threadIdx.y;
    int batch = blockIdx.z;
    if(x_out>=Width_out || y_out >= Height_out)
        return;   
    // We have some nice #defs for you below to simplify indexing. Feel free to use them, or create your own.
    // An example use of these macros:
    // float a = in_4d(0,0,0,0)
    // out_4d(0,0,0,0) = a

    #define out_4d(i3, i2, i1, i0) output[(i3) * (Map_out * Height_out * Width_out) + (i2) * (Height_out * Width_out) + (i1) * (Width_out) + i0]
    #define in_4d(i3, i2, i1, i0) input[(i3) * (Channel * Height * Width) + (i2) * (Height * Width) + (i1) * (Width) + i0]
    #define mask_4d(i3, i2, i1, i0) mask[(i3) * (Channel * K * K) + (i2) * (K * K) + (i1) * (K) + i0]

    // Insert your GPU convolution kernel code here
    float result = 0.0f;
    for(int c=0; c<Channel; c++)
    {
        for(int h=0; h<K; h++)
        {
            for(int w=0; w<K; w++)
            {
                result += in_4d(batch, c, y_out+h, x_out+w)*mask_4d(m, c, h, w);
            }
        }
    }
    out_4d(batch, m, y_out, x_out) = result;

    #undef out_4d
    #undef in_4d
    #undef mask_4d
}

__global__ void unrollAndMatrixMultiplyShared(const float *input_kernel, const float *input_images, float *output_images,
                                     int numARows, int numAColumns,
                                     int numBRows, int numBColumns,
                                     int numCRows, int numCColumns,
                                     int Batch, int K, int Channel, int Height, int Width) {
  #define in_4d(i3, i2, i1, i0) input_images[(i3) * (Channel * Height * Width) + (i2) * (Height * Width) + (i1) * (Width) + i0]

  __shared__ float subTileA[TILE_WIDTH][TILE_WIDTH];
  __shared__ float subTileB[TILE_WIDTH][TILE_WIDTH]; 

  int global_batch = blockIdx.z;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int width_tiles = ceil((1.0*numCColumns)/TILE_WIDTH);
  int row = blockIdx.y*blockDim.y + ty;
  int col = blockIdx.x*blockDim.x + tx; 
  float pval = 0.0f;
  for (int q = 0; q < (numAColumns-1)/TILE_WIDTH + 1; q++) 
  { 
    if ((row < numARows) && (q*TILE_WIDTH+tx < numAColumns))
      subTileA[ty][tx] = input_kernel[row*numAColumns + q*TILE_WIDTH+tx];
    else
      subTileA[ty][tx] = 0.0f;
    
    if ((col < numBColumns) && (q*TILE_WIDTH+ty < numBRows))
    {
      int h_unroll = (q*TILE_WIDTH+ty); // 6
      int w_unroll = col; // 1
      int channel = h_unroll/(K*K); // 1
      int Width_out = Width - K + 1;
      int h = w_unroll/Width_out; // 0
      int w = w_unroll%Width_out; // 1
      int p = (h_unroll - channel*K*K)/K; // (6-4)/2 = 1  
      int _q = (h_unroll - channel*K*K)%K; // (6-4)%2 = 0
      subTileB[ty][tx] = in_4d(global_batch, channel, h+p, w+_q);
    }
    else
      subTileB[ty][tx] = 0.0f;

    __syncthreads();
    for (int k = 0; k < TILE_WIDTH; k++)
      pval += subTileA[ty][k] * subTileB[k][tx];
    __syncthreads();

  }

  if ((row<numCRows)&&(col<numCColumns))
  {
    output_images[global_batch*(numCRows*numCColumns) + row*numCColumns+col] = pval;
  }

  #undef in_4d
}

	
__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // Allocate memory and copy over the relevant data structures to the GPU

    // We pass double pointers for you to initialize the relevant device pointers,
    //  which are passed to the other two functions.

    // Useful snippet for error checking
    // hipError_t error = hipGetLastError();
    // if(error != hipSuccess)
    // {
    //     std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
    //     exit(-1);
    // }
    int Height_out = Height - K + 1;
    int Width_out = Width - K + 1;
    int in_size = Batch*Channel*Height*Width;
    int out_size = Batch*Map_out*Height_out*Width_out;
    int mask_size = Map_out*Channel*K*K;
    hipMalloc((void **)device_input_ptr, in_size*sizeof(float));
    hipMalloc((void **)device_output_ptr, out_size*sizeof(float));
    hipMalloc((void **)device_mask_ptr, mask_size*sizeof(float));
    hipMemcpy(*device_input_ptr, host_input, in_size*sizeof(float), hipMemcpyHostToDevice);
    // hipMemcpy(*device_output_ptr, host_output, out_size*sizeof(float), hipMemcpyHostToDevice);    
    hipMemcpy(*device_mask_ptr, host_mask, mask_size*sizeof(float), hipMemcpyHostToDevice);    

}


__host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // Set the kernel dimensions and call the kernel
    int Height_out = Height - K + 1;
    int Width_out = Width - K + 1;
    int width_tiles = ceil((1.0*Width_out)/TILE_WIDTH);
    int height_tiles = ceil((1.0*Height_out)/TILE_WIDTH);
    if(Channel == 1 && Map_out==4)
    {
        // printf("kernel1");
        int grid_z = Batch;
        int grid_y = width_tiles*height_tiles;
        int grid_x = Map_out;
        dim3 gridDim(grid_x, grid_y, grid_z);
        dim3 blockDim(TILE_WIDTH, TILE_WIDTH, 1);
        conv_forward_kernel<<<gridDim, blockDim>>>(device_output, device_input, device_mask, Batch, Map_out, Channel, Height, Width, K);
    }
    else{
        // printf("kernel2");
        int Width_unroll = Height_out*Width_out;
        int Height_unroll = K*K*Channel;
        int Height_kernel = Map_out;
        int Width_kernel = K*K*Channel;

        int Height_output = Height_kernel;
        int Width_output = Width_unroll;

        int unroll_size = Batch*Height_unroll*Width_unroll;

        dim3 gridDim(ceil(1.0*Width_output/TILE_WIDTH), ceil(1.0*Height_output/TILE_WIDTH), Batch);
        dim3 blockDim(TILE_WIDTH, TILE_WIDTH, 1);

        unrollAndMatrixMultiplyShared<<<gridDim, blockDim>>>(device_mask, device_input, device_output, Height_kernel, Width_kernel, Height_unroll, Width_unroll, Height_output, Width_output, Batch, K, Channel, Height, Width);
    }

 

}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    int Height_out = Height - K + 1;
    int Width_out = Width - K + 1;
    int out_size = Batch*Map_out*Height_out*Width_out;
    // Copy the output back to host
    hipMemcpy(host_output, device_output, out_size*sizeof(float), hipMemcpyDeviceToHost);    

    // Free device memory
    hipFree(device_input);
    hipFree(device_output);
    hipFree(device_mask);

}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}