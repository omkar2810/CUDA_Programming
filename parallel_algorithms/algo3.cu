#include "hip/hip_runtime.h"
// GEMM with shared memory tiling.

#include <wb.h>
#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

#define TILE_WIDTH 16

// Compute C = A * B
__global__ void matrixMultiplyShared(float *A, float *B, float *C,
                                     int numARows, int numAColumns,
                                     int numBRows, int numBColumns,
                                     int numCRows, int numCColumns) {
  //@@ Insert code to implement matrix multiplication here
  //@@ You have to use shared memory for this MP
  __shared__ float subTileA[TILE_WIDTH][TILE_WIDTH];
  __shared__ float subTileB[TILE_WIDTH][TILE_WIDTH]; 

  int bx = blockIdx.x; 
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  int row = by*blockDim.y + ty;
  int col = bx*blockDim.x + tx; 
  float pval = 0;
  for (int q = 0; q < (numAColumns-1)/TILE_WIDTH + 1; q++) 
  { 

    if ((row < numARows) && (q*TILE_WIDTH+tx < numAColumns))
      subTileA[ty][tx] = A[row*numAColumns + q*TILE_WIDTH+tx];
    else
      subTileA[ty][tx] = 0;
    
    if ((col < numBColumns) && (q*TILE_WIDTH+ty < numBRows))
      subTileB[ty][tx] = B[(q*TILE_WIDTH+ty)*numBColumns+col];
    else
      subTileB[ty][tx] = 0;

    __syncthreads();
    for (int k = 0; k < TILE_WIDTH; k++)
      pval += subTileA[ty][k] * subTileB[k][tx];
    __syncthreads();

  }

  if ((row<numCRows)&&(col<numCColumns))
  {
    C[row*numCColumns+col] = pval;
  }
}

int main(int argc, char **argv) {
  wbArg_t args;
  float *hostA; // The A matrix
  float *hostB; // The B matrix
  float *hostC; // The output C matrix

  int numARows;    // number of rows in the matrix A
  int numAColumns; // number of columns in the matrix A
  int numBRows;    // number of rows in the matrix B
  int numBColumns; // number of columns in the matrix B
  int numCRows;    // number of rows in the matrix C (you have to set this)
  int numCColumns; // number of columns in the matrix C (you have to set
                   // this)

  args = wbArg_read(argc, argv);

  //@@ Importing data and creating memory on host
  hostA = (float *)wbImport(wbArg_getInputFile(args, 0), &numARows,
                            &numAColumns);
  hostB = (float *)wbImport(wbArg_getInputFile(args, 1), &numBRows,
                            &numBColumns);

  //@@ Set numCRows and numCColumns
  numCRows = numARows;
  numCColumns = numBColumns;
  int a_size = numARows*numAColumns;
  int b_size = numBRows*numBColumns;
  int c_size = numCRows*numCColumns;

  //@@ Allocate the hostC matrix
  hostC = (float *)malloc(c_size*sizeof(float));

  //@@ Allocate GPU memory here
  float *a_d, *b_d, *c_d;
  hipMalloc((void **) &a_d, a_size*sizeof(float));
  hipMalloc((void **) &b_d, b_size*sizeof(float));
  hipMalloc((void **) &c_d, c_size*sizeof(float));

  //@@ Copy memory to the GPU here
  hipMemcpy(a_d, hostA, a_size*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(b_d, hostB, b_size*sizeof(float), hipMemcpyHostToDevice);


  //@@ Initialize the grid and block dimensions here
  dim3 DimGrid(ceil(1.0*numCColumns/TILE_WIDTH), ceil(1.0*numCRows/TILE_WIDTH), 1);
  dim3 DimBlock(TILE_WIDTH, TILE_WIDTH, 1);

  //@@ Launch the GPU Kernel here
  matrixMultiplyShared<<<DimGrid, DimBlock>>>(a_d, b_d, c_d, numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);
  hipDeviceSynchronize();

  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostC, c_d, c_size*sizeof(float), hipMemcpyDeviceToHost);

  //@@ Free the GPU memory here
  hipFree(a_d);
  hipFree(b_d);
  hipFree(c_d);

  wbSolution(args, hostC, numCRows, numCColumns);

  free(hostA);
  free(hostB);

  //@@ Free the hostC matrix
  free(hostC);

  return 0;
}
